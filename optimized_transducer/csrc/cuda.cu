#include "hip/hip_runtime.h"
// optimized_transducer/csrc/cuda.cu
//
// Copyright (c)  2021  Xiaomi Corporation (authors: Fangjun Kuang)

#include "moderngpu/kernel_load_balance.hxx"
#include "optimized_transducer/csrc/kernels.h"
#include "optimized_transducer/csrc/moderngpu-allocator.h"
#include "torch/script.h"

static constexpr int32_t kMaxThreadsPerBlock = 1024;
static constexpr int32_t kWarpSize = 32;

namespace ot {

static void CheckCuda(hipError_t result, const char *file, int32_t line) {
  if (result != hipSuccess) {
    std::ostringstream os;
    os << file << ":" << line << ": " << hipGetErrorString(result) << "\n";
    throw std::runtime_error(os.str());
  }
}
#define OT_CHECK_CUDA(ret) CheckCuda(ret, __FILE__, __LINE__)

// See https://github.com/k2-fsa/k2/blob/master/k2/csrc/utils.cu#L75
// for the meaning of row splits and row IDs.
/**

  @param row_splits  A 1-D tensor of dtype torch.int32. Its first
                     element should be zero.
  @param num_elems   If -1, it is equal to row_splits[-1].
                     If not -1, it must be equal to row_splits[-1].

  @return Return a 1-D tensor of dtype torch.int32. Its lengths
          equals to num_elems.
 */
torch::Tensor RowSplitsToRowIds(const torch::Tensor &row_splits,
                                int32_t num_elems = -1) {
  torch::CheckedFrom c = "RowSplitsToRowIds";
  auto row_splits_arg = torch::TensorArg(row_splits, "row_splits", 0);
  torch::checkScalarType(c, row_splits_arg, torch::kInt32);
  torch::checkDim(c, row_splits_arg, 1);
  torch::checkContiguous(c, row_splits_arg);

  int32_t num_rows = row_splits.size(0) - 1;
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  if (num_elems == -1) {
    num_elems = row_splits.cpu().data_ptr<int32_t>()[num_rows];
  }

  torch::Tensor row_ids = torch::empty({num_elems}, row_splits.options());
  ModernGpuAllocator allocator;
  mgpu::load_balance_search(num_elems, p_row_splits, num_rows,
                            row_ids.data_ptr<int32_t>(), allocator);
  return row_ids;
}

static std::pair<torch::Tensor, torch::Tensor> ComputeLogProbs(
    const torch::Tensor &logits, const torch::Tensor &denominator,
    const torch::Tensor &targets, const torch::Tensor &logit_lengths,
    const torch::Tensor &target_lengths, int32_t blank) {
  // + 1 here since each sequence is prepended with a blank
  torch::Tensor sizes = logit_lengths * (target_lengths + 1);
  torch::Tensor row_splits = torch::cumsum(sizes, -1, torch::kInt);
  torch::Tensor zero = torch::zeros({1}, row_splits.options());
  row_splits = torch::cat({zero, row_splits}, -1);
  torch::Tensor row_ids = RowSplitsToRowIds(row_splits, logits.size(0));

  const float *p_logits = logits.data_ptr<float>();
  const float *p_den = denominator.data_ptr<float>();
  const int32_t *p_targets = targets.data_ptr<int32_t>();
  const int32_t *p_target_lengths = target_lengths.data_ptr<int32_t>();
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  const int32_t *p_row_ids = row_ids.data_ptr<int32_t>();

  torch::Tensor log_probs = torch::empty({logits.size(0), 2}, logits.options());
  float *p_log_probs = log_probs.data_ptr<float>();

  int32_t num_blocks =
      (logits.size(0) + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;

  ComputeLogProbs<<<num_blocks, kMaxThreadsPerBlock>>>(
      p_logits, p_den, p_targets, p_target_lengths, blank, p_row_splits,
      p_row_ids, logits.size(0), logits.size(1), targets.size(1), p_log_probs);

  auto ret = hipGetLastError();
  OT_CHECK_CUDA(ret);

  return {log_probs, row_splits};
}

static std::pair<torch::Tensor, torch::Tensor> ComputeAlpha(
    const torch::Tensor &log_probs, const torch::Tensor &logit_lengths,
    const torch::Tensor &target_lengths, const torch::Tensor &row_splits) {
  // it is prepended with a blank so we need to use +1 here
  int32_t max_T = logit_lengths.max().item<int32_t>();
  int32_t max_U_p1 = target_lengths.max().item<int32_t>() + 1;
  int32_t batch_size = logit_lengths.size(0);

  int32_t num_warps = (max_T + kWarpSize - 1) / kWarpSize;
  dim3 block_dims(num_warps, max_U_p1, batch_size);
  dim3 thread_dims(kWarpSize);

  torch::Tensor alpha = torch::empty({log_probs.size(0)}, log_probs.options());
  torch::Tensor total_scores = torch::empty({batch_size}, log_probs.options());
  torch::Tensor counter =
      torch::zeros({batch_size * max_U_p1}, logit_lengths.options());

  const float *p_log_probs = log_probs.data_ptr<float>();
  const int32_t *p_logit_lengths = logit_lengths.data_ptr<int32_t>();
  const int32_t *p_target_lengths = target_lengths.data_ptr<int32_t>();
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  int32_t *p_counter = counter.data_ptr<int32_t>();
  float *p_alpha = alpha.data_ptr<float>();
  float *p_total_socres = total_scores.data_ptr<float>();

  ComputeAlpha<<<block_dims, thread_dims>>>(
      p_log_probs, p_logit_lengths, p_target_lengths, p_row_splits, max_T,
      max_U_p1, p_counter, p_alpha, p_total_socres);

  auto ret = hipGetLastError();
  OT_CHECK_CUDA(ret);

  return {alpha, total_scores};
}

std::pair<torch::Tensor, torch::optional<torch::Tensor>>
ComputeTransducerLossCuda(torch::Tensor &logits, const torch::Tensor &targets,
                          const torch::Tensor &logit_lengths,
                          const torch::Tensor &target_lengths, int32_t blank) {
  // The denominator for the log-softmax.
  // Note that it is positive at present.
  torch::Tensor denominator = logits.logsumexp(/*dim*/ 1, /*keepdim*/ false);

  torch::Tensor log_probs;
  torch::Tensor row_splits;

  std::tie(log_probs, row_splits) = ComputeLogProbs(
      logits, denominator, targets, logit_lengths, target_lengths, blank);

  torch::Tensor alpha;
  torch::Tensor total_scores;
  std::tie(alpha, total_scores) =
      ComputeAlpha(log_probs, logit_lengths, target_lengths, row_splits);

  return {total_scores, torch::Tensor()};
}

}  // namespace ot
