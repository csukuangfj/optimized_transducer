#include "hip/hip_runtime.h"
// optimized_transducer/csrc/cuda.cu
//
// Copyright (c)  2021  Xiaomi Corporation (authors: Fangjun Kuang)

#include "moderngpu/kernel_load_balance.hxx"
#include "optimized_transducer/csrc/kernels.h"
#include "optimized_transducer/csrc/moderngpu-allocator.h"
#include "torch/script.h"

static constexpr int32_t kMaxThreadsPerBlock = 1024;
static constexpr int32_t kWarpSize = 32;

namespace ot {

/** Check the status of the return value of some cuda API.

    It throws runtime error exception if the status is not hipSuccess.
 */
static void CheckCuda(hipError_t result, const char *file, int32_t line) {
  if (result != hipSuccess) {
    std::ostringstream os;
    os << file << ":" << line << ": " << hipGetErrorString(result) << "\n";
    throw std::runtime_error(os.str());
  }
}
#define OT_CHECK_CUDA(ret) CheckCuda(ret, __FILE__, __LINE__)

// See https://github.com/k2-fsa/k2/blob/master/k2/csrc/utils.cu#L75
// for the meaning of row splits and row IDs.
/**

  @param row_splits  A 1-D tensor of dtype torch.int32. Its first
                     element should be zero.
  @param num_elems   If -1, it is equal to row_splits[-1].
                     If not -1, it must be equal to row_splits[-1].

  @return Return a 1-D tensor of dtype torch.int32. Its lengths
          equals to num_elems.
 */
torch::Tensor RowSplitsToRowIds(const torch::Tensor &row_splits,
                                int32_t num_elems = -1) {
  torch::CheckedFrom c = "RowSplitsToRowIds";
  auto row_splits_arg = torch::TensorArg(row_splits, "row_splits", 0);
  torch::checkScalarType(c, row_splits_arg, torch::kInt32);
  torch::checkDim(c, row_splits_arg, 1);
  torch::checkContiguous(c, row_splits_arg);

  int32_t num_rows = row_splits.size(0) - 1;
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  if (num_elems == -1) {
    num_elems = row_splits.cpu().data_ptr<int32_t>()[num_rows];
  }

  torch::Tensor row_ids = torch::empty({num_elems}, row_splits.options());
  ModernGpuAllocator allocator;
  mgpu::load_balance_search(num_elems, p_row_splits, num_rows,
                            row_ids.data_ptr<int32_t>(), allocator);
  return row_ids;
}

/**
  @param logits A 2-D tensor of shape (sum_all_TU, vocab_size) containing
                the output from the joint network.
  @param denominator  A 1-D tensor of shape (sum_all_TU,).
  @param targets  A 2-D tensor of shape (batch_size, max_U).
  @param logit_lengths A 1-D tensor of shape (batch_size,)
  @param target_lengths A 1-D tensor of shape (batch_size,)
  @param row_splits A 1-D tensor of shape (batch_size,)
  @param row_ids A 1-D tensor of shape (sum_all_TU,)
  @param blank The ID of the blank symbol.
 */
static torch::Tensor ComputeLogProbs(
    const torch::Tensor &logits, const torch::Tensor &denominator,
    const torch::Tensor &targets, const torch::Tensor &logit_lengths,
    const torch::Tensor &target_lengths, const torch::Tensor &row_splits,
    const torch::Tensor &row_ids, int32_t blank) {
  const float *p_logits = logits.data_ptr<float>();
  const float *p_den = denominator.data_ptr<float>();
  const int32_t *p_targets = targets.data_ptr<int32_t>();
  const int32_t *p_target_lengths = target_lengths.data_ptr<int32_t>();
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  const int32_t *p_row_ids = row_ids.data_ptr<int32_t>();

  torch::Tensor log_probs = torch::empty({logits.size(0), 2}, logits.options());
  float *p_log_probs = log_probs.data_ptr<float>();

  int32_t num_blocks =
      (logits.size(0) + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;

  ComputeLogProbs<<<num_blocks, kMaxThreadsPerBlock, 0,
                    c10::cuda::getCurrentCUDAStream()>>>(
      p_logits, p_den, p_targets, p_target_lengths, blank, p_row_splits,
      p_row_ids, logits.size(0), logits.size(1), targets.size(1), p_log_probs);

  auto ret = hipGetLastError();
  OT_CHECK_CUDA(ret);

  return log_probs;
}

/**
  @param log_probs  A 2-D tensor of shape (sum_all_TU, 2).
  @param logit_lengths A 1-D tensor of shape (batch_size,)
  @param target_lengths A 1-D tensor of shape (batch_size,)
  @param row_splits A 1-D tensor of shape (batch_size,)

  @return Return a pair containing:
    - alpha, a 1-D tensor of shape (sum_all_TU, )
    - total_scores, a 1-D tensor of shape (batch_size,)
 */
static std::pair<torch::Tensor, torch::Tensor> ComputeAlpha(
    const torch::Tensor &log_probs, const torch::Tensor &logit_lengths,
    const torch::Tensor &target_lengths, const torch::Tensor &row_splits) {
  int32_t max_T = logit_lengths.max().item<int32_t>();

  // it is prepended with a blank so we need to use +1 here
  int32_t max_U_p1 = target_lengths.max().item<int32_t>() + 1;

  int32_t batch_size = logit_lengths.size(0);

  int32_t num_warps = (max_T + kWarpSize - 1) / kWarpSize;
  dim3 block_dims(num_warps, max_U_p1, batch_size);
  dim3 thread_dims(kWarpSize);

  torch::Tensor alpha = torch::empty({log_probs.size(0)}, log_probs.options());
  torch::Tensor total_scores = torch::empty({batch_size}, log_probs.options());
  torch::Tensor counter =
      torch::zeros({batch_size * max_U_p1}, logit_lengths.options());

  const float *p_log_probs = log_probs.data_ptr<float>();
  const int32_t *p_logit_lengths = logit_lengths.data_ptr<int32_t>();
  const int32_t *p_target_lengths = target_lengths.data_ptr<int32_t>();
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  int32_t *p_counter = counter.data_ptr<int32_t>();
  float *p_alpha = alpha.data_ptr<float>();
  float *p_total_socres = total_scores.data_ptr<float>();

  ComputeAlpha<<<block_dims, thread_dims, 0,
                 c10::cuda::getCurrentCUDAStream()>>>(
      p_log_probs, p_logit_lengths, p_target_lengths, p_row_splits, max_T,
      max_U_p1, p_counter, p_alpha, p_total_socres);

  auto ret = hipGetLastError();
  OT_CHECK_CUDA(ret);

  return {alpha, total_scores};
}

/**
  @param log_probs  A 2-D tensor of shape (sum_all_TU, 2).
  @param logit_lengths A 1-D tensor of shape (batch_size,)
  @param target_lengths A 1-D tensor of shape (batch_size,)
  @param row_splits A 1-D tensor of shape (batch_size,)

  @param Return the computed beta in a 1-D tensor of shape (sum_all_TU,)
 */
static torch::Tensor ComputeBeta(const torch::Tensor &log_probs,
                                 const torch::Tensor &logit_lengths,
                                 const torch::Tensor &target_lengths,
                                 const torch::Tensor &row_splits) {
  int32_t max_T = logit_lengths.max().item<int32_t>();

  // it is prepended with a blank so we need to use +1 here
  int32_t max_U_p1 = target_lengths.max().item<int32_t>() + 1;

  int32_t batch_size = logit_lengths.size(0);

  int32_t num_warps = (max_T + kWarpSize - 1) / kWarpSize;
  dim3 block_dims(num_warps, max_U_p1, batch_size);
  dim3 thread_dims(kWarpSize);

  // torch::Tensor beta = torch::empty({log_probs.size(0)},
  // log_probs.options());
  torch::Tensor beta = torch::ones({log_probs.size(0)}, log_probs.options());
  torch::Tensor counter =
      torch::zeros({batch_size * max_U_p1}, logit_lengths.options());

  const float *p_log_probs = log_probs.data_ptr<float>();
  const int32_t *p_logit_lengths = logit_lengths.data_ptr<int32_t>();
  const int32_t *p_target_lengths = target_lengths.data_ptr<int32_t>();
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  int32_t *p_counter = counter.data_ptr<int32_t>();
  float *p_beta = beta.data_ptr<float>();

  ComputeBeta<<<block_dims, thread_dims, 0,
                c10::cuda::getCurrentCUDAStream()>>>(
      p_log_probs, p_logit_lengths, p_target_lengths, p_row_splits, max_T,
      max_U_p1, p_counter, p_beta);

  auto ret = hipGetLastError();
  OT_CHECK_CUDA(ret);

  return beta;
}

/**
  @param logits A 2-D tensor of shape (sum_all_TU, vocab_size) containing
                the output from the joint network.
  @param logit_lengths A 1-D tensor of shape (batch_size,)
  @param targets  A 2-D tensor of shape (batch_size, max_U).
  @param target_lengths A 1-D tensor of shape (batch_size,)
  @param denominator  A 1-D tensor of shape (sum_all_TU,).
  @param alpha  A 1-D tensor of shape (sum_all_TU,).
  @param beta  A 1-D tensor of shape (sum_all_TU,).
  @param blank The ID of the blank symbol.
  @param row_splits A 1-D tensor of shape (batch_size,)
  @param row_ids A 1-D tensor of shape (sum_all_TU,)
  @param gradient A 2-D tensor of shape (sum_all_TU, vocab_size).
                  Note: It may share the same underlying memory with
                  `logits`.
 */
static void ComputeGradient(
    const torch::Tensor &logits, const torch::Tensor &logit_lengths,
    const torch::Tensor &targets, const torch::Tensor &target_lengths,
    const torch::Tensor &denominator, const torch::Tensor &alpha,
    const torch::Tensor &beta, int32_t blank, const torch::Tensor &row_splits,
    const torch::Tensor &row_ids, torch::Tensor *gradient) {
  const float *p_logits = logits.data_ptr<float>();
  const int32_t *p_logit_lengths = logit_lengths.data_ptr<int32_t>();
  const int32_t *p_targets = targets.data_ptr<int32_t>();
  const int32_t *p_target_lengths = target_lengths.data_ptr<int32_t>();
  const float *p_den = denominator.data_ptr<float>();
  const float *p_alpha = alpha.data_ptr<float>();
  const float *p_beta = beta.data_ptr<float>();
  const int32_t *p_row_splits = row_splits.data_ptr<int32_t>();
  const int32_t *p_row_ids = row_ids.data_ptr<int32_t>();

  float *p_grad = gradient->data_ptr<float>();

  int32_t num_blocks =
      (logits.size(0) + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;

  ComputeGradient<<<num_blocks, kMaxThreadsPerBlock, 0,
                    c10::cuda::getCurrentCUDAStream()>>>(
      p_logits, p_den, p_targets, p_logit_lengths, p_target_lengths, blank,
      p_row_splits, p_row_ids, logits.size(0), logits.size(1), targets.size(1),
      p_alpha, p_beta, p_grad);

  auto ret = hipGetLastError();
  OT_CHECK_CUDA(ret);
}

std::pair<torch::Tensor, torch::optional<torch::Tensor>>
ComputeTransducerLossCuda(torch::Tensor &logits,  // NOLINT
                          const torch::Tensor &targets,
                          const torch::Tensor &logit_lengths,
                          const torch::Tensor &target_lengths, int32_t blank) {
  torch::DeviceGuard device_guard(logits.device());
  // The denominator for the log-softmax.
  // Note that it is positive at present.
  torch::Tensor denominator = logits.logsumexp(/*dim*/ 1, /*keepdim*/ false);

  // + 1 here since each sequence is prepended with a blank
  torch::Tensor sizes = logit_lengths * (target_lengths + 1);
  torch::Tensor row_splits = torch::cumsum(sizes, -1, torch::kInt);
  torch::Tensor zero = torch::zeros({1}, row_splits.options());
  row_splits = torch::cat({zero, row_splits}, -1);
  torch::Tensor row_ids = RowSplitsToRowIds(row_splits, logits.size(0));

  torch::Tensor log_probs =
      ComputeLogProbs(logits, denominator, targets, logit_lengths,
                      target_lengths, row_splits, row_ids, blank);
  torch::Tensor alpha;
  torch::Tensor total_scores;
  std::tie(alpha, total_scores) =
      ComputeAlpha(log_probs, logit_lengths, target_lengths, row_splits);

  torch::Tensor beta =
      ComputeBeta(log_probs, logit_lengths, target_lengths, row_splits);

  bool requires_grad = logits.requires_grad();
  if (requires_grad) {
    torch::Tensor &gradient = logits;
    ComputeGradient(logits, logit_lengths, targets, target_lengths, denominator,
                    alpha, beta, blank, row_splits, row_ids, &gradient);
  }

  return {total_scores, requires_grad ? logits : torch::Tensor()};
}

}  // namespace ot
