
#include <hip/hip_runtime.h>
// optimized_transducer/csrc/kernels.cu
//
// Copyright (c)  2021  Xiaomi Corporation (authors: Fangjun Kuang)

#define kBlankCol 0
#define kSymCol 1

namespace ot {

__global__ void ComputeLogProbs(const float *logits, const float *denominator,
                                const int32_t *targets,
                                const int32_t *target_lengths, int32_t blank,
                                const int32_t *row_splits,
                                const int32_t *row_ids, int32_t sum_all_TU,
                                int32_t vocab_size, int32_t targets_col,
                                float *log_probs) {
  int32_t idx01 = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx01 >= sum_all_TU) return;  // out-of-boundary

  int32_t b = row_ids[idx01];  // batch size

  // +1 since it is prepended with a blank
  int32_t U_p1 = target_lengths[b] + 1;
  int32_t offset = row_splits[b];
  int32_t idx1 = idx01 - offset;

  int32_t u = idx1 % U_p1;

  const float *p_logits = logits + idx01 * vocab_size;
  const float *p_denominator = denominator + idx01;
  const int32_t *p_targets = targets + b * targets_col;

  float d = *p_denominator;

  float *p_log_probs = log_probs + idx01 * 2;
  p_log_probs[kBlankCol] = p_logits[blank] - d;
  if (u < U_p1 - 1) {
    p_log_probs[kSymCol] = p_logits[p_targets[u]] - d;
  }
}

}  // namespace ot
